#include "hip/hip_runtime.h"
// ---------------------------------------------------------
// Author: Andy Zeng, Princeton University, 2016
// ---------------------------------------------------------

#include <iostream>
#include <fstream>
#include <iomanip>
#include <sstream>
#include <string>
#include "utils.hpp"
#include "hip/hip_vector_types.h"

__device__
float3 matrix_mul(float* K, float3 v) {
  float tmp_pt[3] = {0};
  tmp_pt[0] = v.x - K[0 * 4 + 3];
  tmp_pt[1] = v.y - K[1 * 4 + 3];
  tmp_pt[2] = v.z - K[2 * 4 + 3];
  float x = K[0 * 4 + 0] * tmp_pt[0] + K[1 * 4 + 0] * tmp_pt[1] + K[2 * 4 + 0] * tmp_pt[2];
  float y = K[0 * 4 + 1] * tmp_pt[0] + K[1 * 4 + 1] * tmp_pt[1] + K[2 * 4 + 1] * tmp_pt[2];
  float z = K[0 * 4 + 2] * tmp_pt[0] + K[1 * 4 + 2] * tmp_pt[1] + K[2 * 4 + 2] * tmp_pt[2];
  return make_float3(x, y, z);
}

__device__
bool test_segment(float i, int r0, int r1) {
  return i * 255 > r0 - 0.002 && i * 255 < r1 + 0.002;
}

// CUDA kernel function to integrate a TSDF voxel volume given depth images
__global__
void Integrate(float * cam_K, float * instance_K, float * cam2base, float * cam2world, hipTextureObject_t depth_tex, hipTextureObject_t instance_tex,
               int instance_im_height, int instance_im_width,
               int im_height, int im_width, int3 voxel_grid_dim,
               float3 voxel_grid_origin, float voxel_size, float trunc_margin,
               float * voxel_grid_TSDF, float * voxel_grid_weight, char * voxel_grid_occupancy) {

  int3 pt_grid = make_int3(0, blockIdx.x, threadIdx.x);

  for (int pt_grid_x = 0; pt_grid_x < voxel_grid_dim.x; ++pt_grid_x) {
    pt_grid.x = (float) pt_grid_x;

    float3 pt_base = voxel_grid_origin + make_float3(pt_grid) * voxel_size;

    // Convert from base frame camera coordinates to current frame camera coordinates
    float3 pt_cam = matrix_mul(cam2base, pt_base);

    if (pt_cam.z <= 0)
      continue;

    float pt_pix_x = (cam_K[0 * 3 + 0] * (pt_cam.x / pt_cam.z) + cam_K[0 * 3 + 2]);
    float pt_pix_y = (cam_K[1 * 3 + 1] * (pt_cam.y / pt_cam.z) + cam_K[1 * 3 + 2]);
    if (pt_pix_x < 0 || pt_pix_x >= im_width || pt_pix_y < 0 || pt_pix_y >= im_height)
      continue;

    float pt_ins_x = (instance_K[0 * 3 + 0] * (pt_cam.x / pt_cam.z) + instance_K[0 * 3 + 2]);
    float pt_ins_y = (instance_K[1 * 3 + 1] * (pt_cam.y / pt_cam.z) + instance_K[1 * 3 + 2]);

    float depth_val = tex2D<float>(depth_tex, pt_pix_x / (float) im_width, pt_pix_y / (float) im_height);
    float segment = tex2D<float>(instance_tex, pt_ins_x / (float) instance_im_width, pt_ins_y / (float) instance_im_height);

    float diff = depth_val - pt_cam.z;

    int volume_idx = pt_grid.z * voxel_grid_dim.y * voxel_grid_dim.x + pt_grid.y * voxel_grid_dim.x + pt_grid.x;

    if (diff > 0)
      voxel_grid_occupancy[volume_idx] = FREE;

    if (depth_val <= 0 || depth_val > 6)
    continue;

    if (diff <= -trunc_margin)
      continue;

    // Integrate
    float dist = fmin(1.0f, diff / trunc_margin);
    float weight_old = voxel_grid_weight[volume_idx];
    float weight_new = weight_old + 1.0f;
    voxel_grid_weight[volume_idx] = weight_new;
    voxel_grid_TSDF[volume_idx] = (voxel_grid_TSDF[volume_idx] * weight_old + dist) / weight_new;

    if (std::abs(voxel_grid_TSDF[volume_idx]) < voxel_size / trunc_margin * (0.5 * 1.414) && weight_new > 0.0 && (
      test_segment(segment, 0, 0) ||
      test_segment(segment, 15, 16) ||
      test_segment(segment, 20, 22)
    )) {
      voxel_grid_occupancy[volume_idx] = OCCUPIED;
    }
  }
}

// Loads a binary file with depth data and generates a TSDF voxel volume (5m x 5m x 5m at 1cm resolution)
// Volume is aligned with respect to the camera coordinates of the first frame (a.k.a. base frame)
int main(int argc, char * argv[]) {

  // Location of camera intrinsic file
  std::string cam_K_file = "data/camera-intrinsics.txt";
  std::string instance_K_file = "data/instance-intrinsics.txt";

  // Location of folder containing RGB-D frames and camera pose files
  std::string data_path = "data/rgbd-frames";
  int base_frame_idx = 150;
  int first_frame_idx = 150;
  float num_frames = 50;

  float cam_K[3 * 3];
  float instance_K[3 * 3];
  float base2world[4 * 4];
  float cam2base[4 * 4];
  float cam2world[4 * 4];
  float cam2world_inv[4 * 4];
  int im_width = 640;
  int im_height = 480;
  float depth_im[im_height * im_width];
  int instance_im_width = 1296;
  int instance_im_height = 968;
  float instance_im[instance_im_height * instance_im_width];

  // Voxel grid parameters (change these to change voxel grid resolution, etc.)
  float voxel_grid_origin_x = -1.5f; // Location of voxel grid origin in base frame camera coordinates
  float voxel_grid_origin_y = -1.5f;
  float voxel_grid_origin_z = 0.5f;
  float voxel_size = 0.006f;
  float trunc_margin = voxel_size * 5;
  int voxel_grid_dim_x = 500;
  int voxel_grid_dim_y = 500;
  int voxel_grid_dim_z = 500;

  // Manual parameters
  if (argc > 1) {
    cam_K_file = argv[1];
    data_path = argv[2];
    base_frame_idx = atoi(argv[3]);
    first_frame_idx = atoi(argv[4]);
    num_frames = atof(argv[5]);
    voxel_grid_origin_x = atof(argv[6]);
    voxel_grid_origin_y = atof(argv[7]);
    voxel_grid_origin_z = atof(argv[8]);
    voxel_size = atof(argv[9]);
    trunc_margin = atof(argv[10]);
    instance_K_file = argv[11];
  }

  std::cout << cam_K_file << " " << data_path << " " << base_frame_idx << " " << first_frame_idx << " " << num_frames << std::endl;

  // Read camera intrinsics
  std::vector<float> cam_K_vec = LoadMatrixFromFile(cam_K_file, 3, 3);
  std::copy(cam_K_vec.begin(), cam_K_vec.end(), cam_K);

  std::vector<float> instance_K_vec = LoadMatrixFromFile(instance_K_file, 3, 3);
  std::copy(instance_K_vec.begin(), instance_K_vec.end(), instance_K);

  std::cout << "Camera Intrinsics read" << std::endl;

  // Read base frame camera pose
  std::ostringstream base_frame_prefix;
  base_frame_prefix << std::setw(6) << std::setfill('0') << base_frame_idx;
  std::string base2world_file = data_path + "/frame-" + base_frame_prefix.str() + ".pose.txt";
  std::vector<float> base2world_vec = {
    1.0, 0.0, 0.0, 0.0,
    0.0, 1.0, 0.0, 0.0,
    0.0, 0.0, 1.0, 0.0,
    0.0, 0.0, 0.0, 1.0};//LoadMatrixFromFile(base2world_file, 4, 4);
  std::copy(base2world_vec.begin(), base2world_vec.end(), base2world);

  std::cout << "Base frame pose read" << std::endl;

  // Invert base frame camera pose to get world-to-base frame transform 
  float base2world_inv[16] = {0};
  invert_matrix(base2world, base2world_inv);

  // Initialize voxel grid
  float * voxel_grid_TSDF = new float[voxel_grid_dim_x * voxel_grid_dim_y * voxel_grid_dim_z];
  float * voxel_grid_weight = new float[voxel_grid_dim_x * voxel_grid_dim_y * voxel_grid_dim_z];
  char * voxel_grid_occupancy = new char[voxel_grid_dim_x * voxel_grid_dim_y * voxel_grid_dim_z];
  for (int i = 0; i < voxel_grid_dim_x * voxel_grid_dim_y * voxel_grid_dim_z; ++i)
    voxel_grid_TSDF[i] = 1.0f;
  memset(voxel_grid_weight, 0, sizeof(float) * voxel_grid_dim_x * voxel_grid_dim_y * voxel_grid_dim_z);
  memset(voxel_grid_occupancy, UNKNOWN, sizeof(char) * voxel_grid_dim_x * voxel_grid_dim_y * voxel_grid_dim_z);

  // Load variables to GPU memory
  float * gpu_voxel_grid_TSDF;
  float * gpu_voxel_grid_weight;
  char * gpu_voxel_grid_occupancy;
  hipMalloc(&gpu_voxel_grid_TSDF, voxel_grid_dim_x * voxel_grid_dim_y * voxel_grid_dim_z * sizeof(float));
  hipMalloc(&gpu_voxel_grid_weight, voxel_grid_dim_x * voxel_grid_dim_y * voxel_grid_dim_z * sizeof(float));
  hipMalloc(&gpu_voxel_grid_occupancy, voxel_grid_dim_x * voxel_grid_dim_y * voxel_grid_dim_z * sizeof(char));
  checkCUDA(__LINE__, hipGetLastError());
  hipMemcpy(gpu_voxel_grid_TSDF, voxel_grid_TSDF, voxel_grid_dim_x * voxel_grid_dim_y * voxel_grid_dim_z * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(gpu_voxel_grid_weight, voxel_grid_weight, voxel_grid_dim_x * voxel_grid_dim_y * voxel_grid_dim_z * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(gpu_voxel_grid_occupancy, voxel_grid_occupancy, voxel_grid_dim_x * voxel_grid_dim_y * voxel_grid_dim_z * sizeof(char), hipMemcpyHostToDevice);
  checkCUDA(__LINE__, hipGetLastError());
  float * gpu_cam_K;
  float * gpu_instance_K;
  float * gpu_cam2base;
  float * gpu_cam2world;
  hipMalloc(&gpu_cam_K, 3 * 3 * sizeof(float));
  hipMemcpy(gpu_cam_K, cam_K, 3 * 3 * sizeof(float), hipMemcpyHostToDevice);
  hipMalloc(&gpu_instance_K, 3 * 3 * sizeof(float));
  hipMemcpy(gpu_instance_K, instance_K, 3 * 3 * sizeof(float), hipMemcpyHostToDevice);
  hipMalloc(&gpu_cam2base, 4 * 4 * sizeof(float));
  hipMalloc(&gpu_cam2world, 4 * 4 * sizeof(float));
  checkCUDA(__LINE__, hipGetLastError());

  // Allocate array and copy image data
  hipChannelFormatDesc channelDesc =
  hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
  hipArray *cuArray;
  hipMallocArray(&cuArray, &channelDesc, im_width, im_height);

  hipTextureObject_t         tex;
  hipResourceDesc            texRes;
  memset(&texRes,0,sizeof(hipResourceDesc));

  texRes.resType            = hipResourceTypeArray;
  texRes.res.array.array    = cuArray;

  hipTextureDesc             texDescr;
  memset(&texDescr,0,sizeof(hipTextureDesc));

  texDescr.normalizedCoords = true;
  texDescr.filterMode       = hipFilterModeLinear;
  texDescr.addressMode[0] = hipAddressModeClamp;
  texDescr.addressMode[1] = hipAddressModeClamp;
  texDescr.readMode = hipReadModeElementType;

  hipCreateTextureObject(&tex, &texRes, &texDescr, NULL);

  hipChannelFormatDesc instanceChannelDesc =
  hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
  hipArray *cuArrayInstance;
  hipMallocArray(&cuArrayInstance, &instanceChannelDesc, instance_im_width, instance_im_height);

  hipTextureObject_t         insTex;
  hipResourceDesc            insTexRes;
  memset(&insTexRes,0,sizeof(hipResourceDesc));

  insTexRes.resType            = hipResourceTypeArray;
  insTexRes.res.array.array    = cuArrayInstance;

  hipTextureDesc             instanceTexDescr;
  memset(&instanceTexDescr,0,sizeof(hipTextureDesc));

  instanceTexDescr.normalizedCoords = true;
  instanceTexDescr.filterMode       = hipFilterModeLinear;
  instanceTexDescr.addressMode[0] = hipAddressModeClamp;
  instanceTexDescr.addressMode[1] = hipAddressModeClamp;
  instanceTexDescr.readMode = hipReadModeElementType;

  hipCreateTextureObject(&insTex, &insTexRes, &instanceTexDescr, NULL);

  // Loop through each depth frame and integrate TSDF voxel grid
  for (int frame_idx = first_frame_idx; frame_idx < first_frame_idx + (int)num_frames; ++frame_idx) {

    std::ostringstream curr_frame_prefix;
    curr_frame_prefix << std::setw(6) << std::setfill('0') << frame_idx;

    // // Read current frame depth
    std::string depth_im_file = data_path + "/frame-" + curr_frame_prefix.str() + ".depth.png";
    ReadDepth(depth_im_file, im_height, im_width, depth_im);

    std::string instance_im_file = data_path + "/frame-" + curr_frame_prefix.str() + ".instance.png";
    ReadDepth(instance_im_file, instance_im_height, instance_im_width, instance_im);

    // Read base frame camera pose
    std::string cam2world_file = data_path + "/frame-" + curr_frame_prefix.str() + ".pose.txt";
    std::vector<float> cam2world_vec = LoadMatrixFromFile(cam2world_file, 4, 4);
    std::copy(cam2world_vec.begin(), cam2world_vec.end(), cam2world);

    // Compute relative camera pose (camera-to-base frame)
    multiply_matrix(base2world_inv, cam2world, cam2base);

    invert_matrix(cam2world, cam2world_inv);

    hipMemcpy(gpu_cam2base, cam2base, 4 * 4 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(gpu_cam2world, cam2world_inv, 4 * 4 * sizeof(float), hipMemcpyHostToDevice);
    checkCUDA(__LINE__, hipGetLastError());

    hipMemcpy2DToArray(cuArray, 0, 0, depth_im, im_width * sizeof(float), im_width * sizeof(float), im_height, hipMemcpyHostToDevice);
    hipMemcpy2DToArray(cuArrayInstance, 0, 0, instance_im, instance_im_width * sizeof(float), instance_im_width * sizeof(float), instance_im_height, hipMemcpyHostToDevice);

    std::cout << "Fusing: " << depth_im_file << std::endl;

    Integrate <<< voxel_grid_dim_z, voxel_grid_dim_y >>> (gpu_cam_K, gpu_instance_K, gpu_cam2base, gpu_cam2world, tex, insTex, instance_im_height, instance_im_width,
                                                          im_height, im_width, make_int3(voxel_grid_dim_x, voxel_grid_dim_y, voxel_grid_dim_z),
                                                          make_float3(voxel_grid_origin_x, voxel_grid_origin_y, voxel_grid_origin_z), voxel_size, trunc_margin,
                                                          gpu_voxel_grid_TSDF, gpu_voxel_grid_weight, gpu_voxel_grid_occupancy);
  }

  // Load TSDF voxel grid from GPU to CPU memory
  hipMemcpy(voxel_grid_TSDF, gpu_voxel_grid_TSDF, voxel_grid_dim_x * voxel_grid_dim_y * voxel_grid_dim_z * sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(voxel_grid_weight, gpu_voxel_grid_weight, voxel_grid_dim_x * voxel_grid_dim_y * voxel_grid_dim_z * sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(voxel_grid_occupancy, gpu_voxel_grid_occupancy, voxel_grid_dim_x * voxel_grid_dim_y * voxel_grid_dim_z * sizeof(bool), hipMemcpyDeviceToHost);
  checkCUDA(__LINE__, hipGetLastError());

  // Compute surface points from TSDF voxel grid and save to point cloud .ply file
  std::cout << "Saving surface point cloud (tsdf.ply)..." << std::endl;
  SaveVoxelGrid2SurfacePointCloud("tsdf.ply", voxel_grid_dim_x, voxel_grid_dim_y, voxel_grid_dim_z, 
                                  voxel_size, voxel_grid_origin_x, voxel_grid_origin_y, voxel_grid_origin_z,
                                  voxel_grid_TSDF, voxel_grid_weight, voxel_grid_occupancy, 0.2f, 0.0f);

  {
    // Save TSDF voxel grid and its parameters to disk as binary file (float array)
    std::cout << "Saving TSDF voxel grid values to disk (tsdf.bin)..." << std::endl;
    std::string voxel_grid_saveto_path = "tsdf.bin";
    std::ofstream outFile(voxel_grid_saveto_path, std::ios::binary | std::ios::out);
    float voxel_grid_dim_xf = (float) voxel_grid_dim_x;
    float voxel_grid_dim_yf = (float) voxel_grid_dim_y;
    float voxel_grid_dim_zf = (float) voxel_grid_dim_z;
    outFile.write((char*)&voxel_grid_dim_xf, sizeof(float));
    outFile.write((char*)&voxel_grid_dim_yf, sizeof(float));
    outFile.write((char*)&voxel_grid_dim_zf, sizeof(float));
    outFile.write((char*)&voxel_grid_origin_x, sizeof(float));
    outFile.write((char*)&voxel_grid_origin_y, sizeof(float));
    outFile.write((char*)&voxel_grid_origin_z, sizeof(float));
    outFile.write((char*)&voxel_size, sizeof(float));
    outFile.write((char*)&trunc_margin, sizeof(float));
    for (int i = 0; i < voxel_grid_dim_x * voxel_grid_dim_y * voxel_grid_dim_z; ++i)
      outFile.write((char*)&voxel_grid_TSDF[i], sizeof(float));
    outFile.close();
  }

  // Save voxel occupancy grid and its parameters to disk as binary file (float array)
  {
    std::cout << "Saving voxel occupancy grid values to disk (occupancy.bin)..." << std::endl;
    std::string voxel_grid_saveto_path = "occupancy.bin";
    std::ofstream outFile(voxel_grid_saveto_path, std::ios::binary | std::ios::out);
    for (int i = 0; i < voxel_grid_dim_x * voxel_grid_dim_y * voxel_grid_dim_z; ++i)
      outFile.write((char*)&voxel_grid_occupancy[i], sizeof(char));
    outFile.close();
  }

  return 0;
}


